#include "hip/hip_runtime.h"
// adapted from https://github.com/NVIDIA/FasterTransformer

#include <ATen/cuda/HIPContext.h>
#include <hiprand/hiprand_kernel.h>
#include <glog/logging.h>
#include <torch/torch.h>

#include <hipcub/hipcub.hpp>

#include "../dispatch.h"
#include "../reduce_kernel_utils.cuh"

namespace llm::kernel {

// reduce topk for each thread block and save the result to temp storage
template <typename T, int BLOCK_SIZE, int BLOCKS_PER_SEQ>
__global__ void partial_topk_within_block(const T* __restrict logits,
                                          T* __restrict tmp_logits,
                                          int* __restrict tmp_topk_ids,
                                          T* __restrict tmp_topk_logits,
                                          int max_top_k,
                                          const int* __restrict top_ks,
                                          int vocab_size) {
  typedef hipcub::BlockReduce<TopK_2<T>, BLOCK_SIZE> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;

  // one sequence is splited into BLOCKS_PER_SEQ blocks for parallel processing
  const int batch_id = bid / BLOCKS_PER_SEQ;
  // block lane for the sequence
  const int block_lane = bid % BLOCKS_PER_SEQ;
  const int k = top_ks[batch_id];

  const int tmp_logits_base = batch_id * vocab_size;
  const int tmp_topk_base =
      batch_id * BLOCKS_PER_SEQ * max_top_k + block_lane * k;

  // copy log_probs to tmp_log_probs for modifying
#pragma unroll
  for (int id = tid + block_lane * BLOCK_SIZE; id < vocab_size;
       id += BLOCK_SIZE * BLOCKS_PER_SEQ) {
    const int index = id + tmp_logits_base;
    tmp_logits[index] = logits[index];
  }

  TopK_2<T> partial;
  const T MAX_T_VAL = (std::is_same_v<T, half>) ? HALF_FLT_MAX : FLT_MAX;

  // every thread does top-k then combine the result with block reduce
  for (int ite = 0; ite < k; ite++) {
    partial.init();

#pragma unroll
    for (int id = tid + block_lane * BLOCK_SIZE; id < vocab_size;
         id += BLOCK_SIZE * BLOCKS_PER_SEQ) {
      const int index = id + tmp_logits_base;
      partial.insert(tmp_logits[index], index);
    }

    // reduce within each block
    TopK_2<T> total =
        BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<T>);

    // save the topk idx and value to temp storage
    if (tid == 0) {
      const int index = tmp_topk_base + ite;
      tmp_topk_ids[index] = total.p;
      tmp_topk_logits[index] = total.u;
      // remove the largest item by setting the score to -MAX_T_VAL
      tmp_logits[total.p] = -MAX_T_VAL;
    }

    // wait for all threads to finish
    __syncthreads();
  }
}

// reduce topk across blocks for each batch
template <typename T, int BLOCK_SIZE, int BLOCKS_PER_SEQ>
__global__ void topk_sampling_across_blocks(int* output_ids,
                                            float* output_log_probs,
                                            const int* __restrict tmp_topk_ids,
                                            T* __restrict tmp_topk_logits,
                                            int max_top_k,
                                            const int* __restrict top_ks,
                                            const float* __restrict top_ps,
                                            hiprandState_t* curandstate) {
  typedef hipcub::BlockReduce<TopK_2<float>, BLOCK_SIZE> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  const T MAX_T_VAL = (std::is_same_v<T, half>) ? HALF_FLT_MAX : FLT_MAX;

  const int tid = threadIdx.x;
  // each block processes one sequence
  const int batch_id = blockIdx.x;
  const int k = top_ks[batch_id];
  const float p = top_ps != nullptr ? top_ps[batch_id] : 1.0f;
  const int stride = max_top_k * BLOCKS_PER_SEQ;

  // move the pointer to the corresponding batch
  T* topk_logits = tmp_topk_logits + batch_id * stride;
  const int* topk_ids = tmp_topk_ids + batch_id * stride;

  // use shared memory to save temp topk idxs and values
  extern __shared__ char smem[];  // idxs + vals for topk
  int* s_idxs = reinterpret_cast<int*>(smem);
  float* s_vals = reinterpret_cast<float*>(s_idxs + k);

  // use shared memory to save sum and max value for softmax
  __shared__ float s_sum_val;
  __shared__ float s_max_val;
  if (tid == 0) {
    // add a small epsilon to avoid division by zero
    s_sum_val = 1e-6f;
  }

  // each block has a partial topk
  const int size = k * BLOCKS_PER_SEQ;
  // use float to record laggest value
  TopK_2<float> partial;
  // calculate topk and softmax for each sequence
  for (int ite = 0; ite < k; ++ite) {
    partial.init();

    // merge partial topk from all blocks
#pragma unroll
    for (int i = tid; i < size; i += BLOCK_SIZE) {
      partial.insert(topk_logits[i], i);
    }

    // reduce within each block to get the top idx and value
    TopK_2<float> total =
        BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<float>);

    // save the topk idx and value to shared memory
    if (tid == 0) {
      if (ite == 0) {
        s_max_val = total.u;
      }
      s_idxs[ite] = total.p;
      // remove the largest item by setting the score to -MAX_T_VAL
      topk_logits[total.p] = -MAX_T_VAL;

      // calculate expf(x - max_val) and sum for softmax
      const float exp_logit = __expf(total.u - s_max_val);
      s_vals[ite] = exp_logit;
      s_sum_val += exp_logit;
    }
    __syncthreads();
  }

  // let thread 0 sample the id from topk candidates
  if (tid == 0) {
    float rand_num = hiprand_uniform(curandstate + batch_id) * p * s_sum_val;
    for (int i = 0; i < k; ++i) {
      const float exp_logit = s_vals[i];
      rand_num -= exp_logit;
      if (rand_num <= 0 || i == k - 1) {
        output_ids[batch_id] = topk_ids[s_idxs[i]];
        // the log_prob is the probability of the selected tokens
        const float log_prob = logf(exp_logit / s_sum_val);
        output_log_probs[batch_id] = log_prob;
        break;
      }
    }
  }
}

// topk sampling kernel launcher that calculates the topk for each sequence in
// following steps:
// 1. split the sequence into BLOCKS_PER_SEQ blocks for parallel processing and
// calculate the partial topk for each block
// 2. reduce the partial topk across blocks for each sequence
#define CASE_K_RANGE(K_MIN, K_MAX, BLOCK_SIZE_1, BLOCK_SIZE_2, BLOCKS_PER_SEQ) \
  case K_MIN ... K_MAX:                                                        \
    partial_topk_within_block<scalar_t, BLOCK_SIZE_1, BLOCKS_PER_SEQ>          \
        <<<batch_size * BLOCKS_PER_SEQ, BLOCK_SIZE_1, 0, stream>>>(            \
            _logits,                                                           \
            tmp_logits,                                                        \
            tmp_topk_ids,                                                      \
            tmp_topk_logits,                                                   \
            max_top_k,                                                         \
            _top_ks,                                                           \
            vocab_size);                                                       \
    topk_sampling_across_blocks<scalar_t, BLOCK_SIZE_2, BLOCKS_PER_SEQ>        \
        <<<batch_size,                                                         \
           BLOCK_SIZE_2,                                                       \
           K_MAX * sizeof(int) + K_MAX * sizeof(float),                        \
           stream>>>(_output_ids,                                              \
                     _output_log_probs,                                        \
                     tmp_topk_ids,                                             \
                     tmp_topk_logits,                                          \
                     max_top_k,                                                \
                     _top_ks,                                                  \
                     _top_ps,                                                  \
                     curandstate);                                             \
    break;

void invoke_topk_sampling(torch::Tensor& output_ids,
                          torch::Tensor& output_log_probs,
                          torch::Tensor logits,
                          torch::Tensor workspace,
                          hiprandState_t* curandstate,
                          int max_top_k,
                          torch::Tensor top_ks,
                          torch::Tensor top_ps) {
  const int batch_size = logits.size(0);
  const int vocab_size = logits.size(1);
  const int max_blocks_per_seq = 8;

  // tmp_logits to save modified logits
  size_t tmp_logits_size = batch_size * vocab_size;
  // tmp_topk_* to save topk ids and logits for each block
  size_t tmp_topk_size = batch_size * max_blocks_per_seq * max_top_k;
  // round up to prevent memory misalignment
  tmp_logits_size = ((tmp_logits_size + 3) / 4) * 4;
  tmp_topk_size = ((tmp_topk_size + 3) / 4) * 4;

  DISPATCH_FLOATING_TYPES(logits.scalar_type(), "tok_kernel", [&] {
    const size_t min_workspace_size =
        tmp_logits_size * sizeof(scalar_t) +
        tmp_topk_size * (sizeof(int) + sizeof(scalar_t));
    assert(workspace.numel() >= min_workspace_size);

    // scratch space for topk
    scalar_t* tmp_logits = workspace.data_ptr<scalar_t>();
    int* tmp_topk_ids = reinterpret_cast<int*>(tmp_logits + tmp_logits_size);
    scalar_t* tmp_topk_logits =
        reinterpret_cast<scalar_t*>(tmp_topk_ids + tmp_topk_size);

    int* _output_ids = output_ids.data_ptr<int>();
    float* _output_log_probs = output_log_probs.data_ptr<float>();

    const scalar_t* _logits = logits.data_ptr<scalar_t>();
    const int* _top_ks = top_ks.data_ptr<int>();
    const float* _top_ps =
        top_ps.defined() ? top_ps.data_ptr<float>() : nullptr;

    auto stream = at::cuda::getCurrentCUDAStream();
    switch (max_top_k) {
      // K_MIN, K_MAX, BLOCK_SIZE_1, BLOCK_SIZE_2, BLOCKS_PER_SEQ
      CASE_K_RANGE(1, 16, 128, 128, 8);
      CASE_K_RANGE(17, 32, 256, 128, 8);
      CASE_K_RANGE(33, 64, 256, 256, 8);
      CASE_K_RANGE(65, 1024, 256, 256, 8);
      default:
        LOG(FATAL) << "topk_sampling only supports max_top_k <= 1024 but got "
                   << max_top_k;
    }
  });
}
#undef CASE_K

}  // namespace llm::kernel